//
// Created by herve on 13-04-2024.
//

#include "histogram_eq.h"

namespace cp
{
    constexpr auto HISTOGRAM_LENGTH = 256;

    static float prob(const int x, const int size)
    {
        return static_cast<float>(x) / static_cast<float>(size);
    }


    static unsigned char correct_color(const float cdf_val, const float cdf_min)
    {
        return static_cast<unsigned char>(255 * (cdf_val - cdf_min) / (1 - cdf_min));
    }


    static void histogram_equalization(const int width, const int height,
                                       const float* input_image_data,
                                       float* output_image_data,
                                       const std::shared_ptr<unsigned char[]>& uchar_image,
                                       const std::shared_ptr<unsigned char[]>& gray_image,
                                       int (&histogram)[HISTOGRAM_LENGTH],
                                       float (&cdf)[HISTOGRAM_LENGTH]
    )
    {
        return;
    }

    wbImage_t iterative_histogram_equalization(const wbImage_t& input_image, const int iterations)
    {
        // Input 1 2 5 4 9 7 0 1
        // Output: 1 3 8 12 21 28 28 29

        const auto width = wbImage_getWidth(input_image);
        const auto height = wbImage_getHeight(input_image);
        constexpr auto channels = 3;
        const auto size = width * height;
        const auto size_channels = size * channels;

        const wbImage_t output_image = wbImage_new(width, height, channels);
        float* input_image_data = wbImage_getData(input_image);
        float* output_image_data = wbImage_getData(output_image);

        std::shared_ptr<unsigned char[]> uchar_image(new unsigned char[size_channels]);
        std::shared_ptr<unsigned char[]> gray_image(new unsigned char[size]);

        int histogram[HISTOGRAM_LENGTH];
        float cdf[HISTOGRAM_LENGTH];

        for (int i = 0; i < iterations; i++)
        {
            histogram_equalization(width, height,
                                   input_image_data, output_image_data,
                                   uchar_image, gray_image,
                                   histogram, cdf);

            input_image_data = output_image_data;
        }

        return output_image;
    }
}
